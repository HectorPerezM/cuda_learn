#include "hip/hip_runtime.h"
#include "../common/book.h"

#define N 20

__global__ void add(int *a, int *b, int *c) {
    int tid = threadIdx.x;
    if(tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    //allocate memory on GPU
    HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N*sizeof(int)));

    //fill arrays 'a' and 'b' on CPU
    for(int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    //copy arrays 'a' and 'b' on GPU
    HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice));

    add<<<1,N>>>(dev_a, dev_b, dev_c);

    //copy array 'c' back from GPU to CPU
    HANDLE_ERROR(hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost));

    //display results
    for(int i=0; i<N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    //free mem allocated on gpu
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}