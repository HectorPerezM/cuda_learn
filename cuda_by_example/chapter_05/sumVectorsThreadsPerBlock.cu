#include "hip/hip_runtime.h"
#include "../common/book.h"

#define N (33 * 1024)

__global__ void add(int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while(tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    //allocate memory on GPU
    HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N*sizeof(int)));

    //fill arrays 'a' and 'b' on CPU
    for(int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    //copy arrays 'a' and 'b' on GPU
    HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice));

    add<<<128,128>>>(dev_a, dev_b, dev_c);

    //copy array 'c' back from GPU to CPU
    HANDLE_ERROR(hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost));

    //display results
    bool success = true;
    for(int i=0; i<N; i++) {
        if((a[i] + b[i]) != c[i]) {
            printf("%d + %d = %d\n", a[i], b[i], c[i]);
            success = false;
        }
    }

    if(success) printf("we did it\n");

    //free mem allocated on gpu
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}